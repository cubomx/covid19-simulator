#include "hip/hip_runtime.h"


#include <algorithm>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <stdlib.h> /* srand, rand */
#include <time.h>   /* time */
#include <vector>

using namespace std;

__device__ __host__ int randomInt(int, int);
__device__ __host__ float randomFloat(float, float);

class Agent {
public:
  float contagionProba;        // 0.02 - 0.03
  float extContagionProba;     // 0.02 - 0.03
  float deathProba;            // 0.007 - 0.07
  float movProba;              // 0.3 - 0.5
  float shortDistanceMovProba; // 0.7 - 0.9
  int incubationTime;          // 5 - 6
  int recoveryTime;            // 14

  int status; /* (0) No infected, (1) infected, (-1) Quarantine (-3) Decease
               */
  float posX; // 0 - p
  float posY; // 0 - q

  __device__ __host__ void generate(float x, float y) {
    posX = x;
    posY = y;
    contagionProba = randomFloat(0.02, 0.03);
    extContagionProba = randomFloat(0.02, 0.03);
    deathProba = randomFloat(0.007, 0.07);
    movProba = randomFloat(0.3, 0.5);
    shortDistanceMovProba = randomFloat(0.7, 0.9);
    incubationTime = randomInt(5, 6);
    recoveryTime = 14;
    status = 0;
  }
};

// Quantity of agents
const int numAgents = 10240;
// Days of duration of the simulation
const int numDays = 160;
// Maximum number of movements per day
const int maxNumMovDay = 30;
// Maximum radius of local movement
const int radiusMaxMovLocal = 5;
// Meters of distance that the virus can travel
const int distanceContagion = 1;

int xSize = 500, ySize = 500;

int deaths = 0;

int contagions = 0;

void initAgents(vector<Agent> &agents, Agent *map[][500]) {
  for (int i = 0; i < numAgents; i++) {
    int posX = 0;
    int posY = 0;

    do {
      posX = randomInt(0, 500);
      posY = randomInt(0, 500);
    } while (map[posX][posY] != NULL);

    Agent newAgente = Agent();

    newAgente.generate(posX, posY);

    agents.push_back(newAgente);
    map[posX][posY] = &newAgente;
  }
}

void simulate(vector<Agent> &, Agent *[][500]);
void contagion(Agent *, Agent *[][500]);
void movility(Agent *, Agent *[][500]);
void externContagion(Agent *);
void contagionEffects(Agent *);
void decease(Agent *);

__device__ float generate(hiprandState *globalState, int ind) {
  hiprandState localState = globalState[ind];
  float RANDOM = hiprand_uniform(&localState);
  globalState[ind] = localState;
  return RANDOM;
}

__device__ float generateRand(hiprandState *globalState, int ind, float low,
                              float high) {
  hiprandState localState = globalState[ind];
  float RANDOM = hiprand_uniform(&localState) * (high);
  if (RANDOM < low) {
    RANDOM = low;
  }
  globalState[ind] = localState;
  return RANDOM;
}

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int threadId = blockId * (blockDim.x * blockDim.y) +
                 (threadIdx.y * blockDim.x) + threadIdx.x;
  hiprand_init(seed, threadId, 0, &state[threadId]);
}

__device__ float EuclideanDistance(Agent agent1, Agent Agent2) {
  float suma = 0;
  suma = pow(agent1.posX - Agent2.posX, 2);
  suma += pow(agent1.posY - Agent2.posY, 2);
  return float(sqrt(suma));
}

__global__ void GPU_contagio(Agent *agent, hiprandState *globalState) {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int thisId = blockId * (blockDim.x * blockDim.y) +
               (threadIdx.y * blockDim.x) + threadIdx.x;
  for (int bIdY = 0; bIdY < gridDim.y; bIdY++) {
    for (int bIdX = 0; bIdX < gridDim.x; bIdX++) {
      for (int tIdY = 0; tIdY < blockDim.y; tIdY++) {
        for (int tIdX = 0; tIdX < blockDim.x; tIdX++) {
          int blockId = bIdX + bIdY * gridDim.x;
          int otherId =
              blockId * (blockDim.x * blockDim.y) + (tIdY * blockDim.x) + tIdX;
          if (agent[otherId].status == 1 && agent[thisId].status == 0) {
            if (EuclideanDistance(agent[thisId], agent[otherId]) <= 1.0) {
              if (generate(globalState, thisId) <=
                  agent[thisId].contagionProba) {
                agent[thisId].status = 1;
              }
            }
          }
        }
      }
    }
  }
}

__global__ void GPU_Efects(Agent *agent, hiprandState *globalState) {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int thisId = blockId * (blockDim.x * blockDim.y) +
               (threadIdx.y * blockDim.x) + threadIdx.x;

  if (agent[thisId].status == 1) {
    if (agent[thisId].incubationTime == 0) {
      agent[thisId].status = -1;
    }
    agent[thisId].incubationTime -= 1;

  } else if (agent[thisId].status == -1) {
    if (agent[thisId].recoveryTime > 0) {
      agent[thisId].recoveryTime--;
    }
  }
}

__global__ void GPU_movility(Agent *agent, hiprandState *globalState,
                             int *maxMovementDistance) {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int threadId = blockId * (blockDim.x * blockDim.y) +
                 (threadIdx.y * blockDim.x) + threadIdx.x;

  float xLimit = 500.0, yLimit = 500.0;

  float actualX = agent[threadId].posX, actualY = agent[threadId].posY;
  float movementX = 0.0, movementY = 0.0;
  bool movValido = false;
  if (agent[threadId].status == -1 && agent[threadId].incubationTime == 0) {
    return;
  }
  if (generate(globalState, threadId) <= agent[threadId].movProba) {
    if (generate(globalState, threadId) <=
        agent[threadId].shortDistanceMovProba) {
      do {
        movementX = 2 * generate(globalState, threadId) - 1;
        movementY = 2 * generate(globalState, threadId) - 1;
        if (movementX + actualX >= 0.0 || movementX + actualX < xLimit) {
          if (movementY + actualY >= 0.0 || movementY + actualY < yLimit) {
            if ((movementX * movementX) + (movementY * movementY) <=
                (*maxMovementDistance * *maxMovementDistance)) {
              movValido = true;
            }
          }
        }
      } while (!movValido);
    } else {
      do {
        movementX = xLimit * generate(globalState, threadId);
        movementY = yLimit * generate(globalState, threadId);
        if (movementX + actualX >= 0.0 || movementX + actualX < xLimit) {
          if (movementY + actualY >= 0.0 || movementY + actualY < yLimit) {
            if ((movementX * movementX) + (movementY * movementY) <=
                (*maxMovementDistance * *maxMovementDistance)) {
              movValido = true;
            }
          }
        }
      } while (!movValido);
    }
    agent[threadId].posX = actualX + movementX;
    agent[threadId].posY = actualY + movementY;
  }
}

__global__ void GPU_externContagion(Agent *agent, hiprandState *globalState) {
  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int threadId = blockId * (blockDim.x * blockDim.y) +
                 (threadIdx.y * blockDim.x) + threadIdx.x;
  if (agent[threadId].status == 0) {
    if (generate(globalState, threadId) * 1.0 <=
        agent[threadId].extContagionProba) {
      agent[threadId].status = 1;
    }
  }
}

__global__ void initAgents(Agent *agent, hiprandState *globalState) {

  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
  int threadId = blockId * (blockDim.x * blockDim.y) +
                 (threadIdx.y * blockDim.x) + threadIdx.x;

  float posX = generate(globalState, threadId) * 500.0;
  float posY = generate(globalState, threadId) * 500.0;

  Agent agentNew = Agent();
  agentNew.posX = posX;
  agentNew.posY = posY;

  agentNew.contagionProba = generateRand(globalState, threadId, 0.02, 0.03);
  agentNew.extContagionProba = generateRand(globalState, threadId, 0.02, 0.03);
  agentNew.deathProba = generateRand(globalState, threadId, 0.007, 0.07);
  agentNew.movProba = generateRand(globalState, threadId, 0.3, 0.5);
  agentNew.shortDistanceMovProba =
      generateRand(globalState, threadId, 0.7, 0.9);
  /* Incubation time */
  agentNew.incubationTime = 5;
  if (generate(globalState, threadId) > 0.5) {
    agentNew.incubationTime = 6;
  }
  agentNew.recoveryTime = 14;
  agentNew.status = 0;
  agent[threadId] = agentNew;
}

__host__ int checkCurrentContagions(Agent *agentsCPU) {
  int total = 0;
  for (int i = 0; i < numAgents; i++) {
    if (agentsCPU[i].status == 1 ||
        (agentsCPU[i].status == -1 && agentsCPU[i].recoveryTime > 0)) {
      total++;
    }
  }
  return total;
}

__host__ int checkCurrentRecovered(Agent *agentsCPU) {
  int total = 0;
  for (int i = 0; i < numAgents; i++) {
    if (agentsCPU[i].recoveryTime <= 0) {
      total++;
    }
  }
  return total;
}

__host__ int checkCurrentDeaths(Agent *agentsCPU) {
  int total = 0;
  for (int i = 0; i < numAgents; i++) {
    if (agentsCPU[i].status == -3) {
      total++;
    }
  }
  return total;
}

int main() {

  srand(time(NULL));

  /*static Agent* map[500][500] = { 0 };
  vector<Agent> agents;

  initAgents(agents, map);
  simulate(agents, map);

  cout << "Total sum of contagions in " << numDays << " days is: " << contagions
  << "\n"; cout << "Total sum of deaths in " << numDays << " days is: " <<
  deaths << "\n";
  */

  static Agent agents[numAgents];

  Agent *agentsGPU;
  Agent *agentsCPU;

  const size_t size = size_t(numAgents) * sizeof(Agent);

  agentsCPU = (Agent *)malloc(size);

  hipMalloc((void **)&agentsGPU, size);

  hipMemcpy(agentsGPU, &agents[0], size, hipMemcpyHostToDevice);

  hiprandState *devStates;
  hipMalloc(&devStates, numAgents * sizeof(hiprandState));
  dim3 block(5, 2);
  dim3 grid(32, 32);
  srand(time(0));
  int *devRadiusMaxMovLocal;
  hipMalloc(&devRadiusMaxMovLocal, sizeof(int));
  hipMemcpy(devRadiusMaxMovLocal, &radiusMaxMovLocal, sizeof(int),
             hipMemcpyHostToDevice);
  /* setup the kernel for the random numbers */
  int seed = rand();
  setup_kernel<<<grid, block>>>(devStates, time(NULL));

  initAgents<<<grid, block>>>(agentsGPU, devStates);

  hipMemcpy(agentsCPU, agentsGPU, size, hipMemcpyDeviceToHost);

  /*for (int i = 100; i < 200; i++) {
      printf("%f \n", agentsCPU[i].extContagionProba);
  }*

  /* GPU simulation */

  int day = 0, movement;
  while (day < numDays) {
    movement = 0;
    GPU_externContagion<<<grid, block>>>(agentsGPU, devStates);
    hipDeviceSynchronize();
    hipMemcpy(agentsCPU, agentsGPU, size, hipMemcpyDeviceToHost);
    hipMemcpy(agentsGPU, agentsCPU, size, hipMemcpyHostToDevice);
    while (movement < maxNumMovDay) {
      hipMemcpy(agentsGPU, agentsCPU, size, hipMemcpyHostToDevice);
      GPU_contagio<<<grid, block>>>(agentsGPU, devStates);
      hipDeviceSynchronize();
      hipMemcpy(agentsCPU, agentsGPU, size, hipMemcpyDeviceToHost);
      hipMemcpy(agentsGPU, agentsCPU, size, hipMemcpyHostToDevice);
      GPU_movility<<<grid, block>>>(agentsGPU, devStates, devRadiusMaxMovLocal);

      hipDeviceSynchronize();
      hipMemcpy(agentsCPU, agentsGPU, size, hipMemcpyDeviceToHost);
      movement++;
    }
    hipMemcpy(agentsGPU, agentsCPU, size, hipMemcpyHostToDevice);
    GPU_Efects<<<grid, block>>>(agentsGPU, devStates);
    hipDeviceSynchronize();
    hipMemcpy(agentsCPU, agentsGPU, size, hipMemcpyDeviceToHost);
    hipMemcpy(agentsGPU, agentsCPU, size, hipMemcpyHostToDevice);
    int currentContagions = checkCurrentContagions(agentsCPU);
    int currentDeaths = checkCurrentDeaths(agentsCPU);
    int currentRecovered = checkCurrentRecovered(agentsCPU);
    printf("Day: %d, Current infected: %d, Current  Recovered: %d, Current "
           "Dead: %d.\n",
           day, currentContagions, currentRecovered, currentDeaths);
    day++;
  }
}

/*
    Handles the days simulating the probability of getting the virus
    and the struggles of getting it.
*/

void simulate(vector<Agent> &agents, Agent *map[][500]) {
  int day = 0, movement = 0;
  printf("%llu\n", agents.size());
  while (day < numDays) { // days of the simulation
    movement = 0;

    while (movement < maxNumMovDay) {
      for (auto &agent : agents) {
        contagion(&agent, map);
        movility(&agent, map);
      }
      movement++;
    }
    for (auto &agent : agents) {
      externContagion(&agent);
      contagionEffects(&agent);
      decease(&agent);
    }
    day++;
  }
}

// /*
//     Function to check if there's someone surrounding (when the agent is not
//     infected) and check if they may get infected by some of the agents near
//     to the actual agent and if they currently are infected.
// */

// void contagion(Agent *agent, Agent *map[][500]) {
//   // Check if isn't infected
//   if (agent->status == 0) {
//     int x = agent->posX;
//     int y = agent->posY;
//     int newState = randomFloat(0.0, 1.0);
//     // Check if neighbors to a distance of 1 meter
//     int itGetInfected = 0;
//     if (x + 1 < xSize) {
//       if (map[x + 1][y] != nullptr && map[x + 1][y]->status > 0) {
//         itGetInfected = 1;
//       }
//     }
//     if (x - 1 >= 0) {
//       if (map[x - 1][y] != nullptr && map[x - 1][y]->status > 0) {
//         itGetInfected = 1;
//       }
//     }

//     if (map[x][y + 1] != nullptr && y + 1 < ySize) {
//       if (map[x][y + 1]->status > 0) {
//         itGetInfected = 1;
//       }
//     }

//     if (y - 1 >= 0) {
//       if (map[x][y - 1] != nullptr && map[x][y - 1]->status > 0) {
//         itGetInfected = 1;
//       }
//     }

//     if (newState <= agent->contagionProba && itGetInfected == 1) {
//       agent->status = 1;
//       contagions++;
//     }
//   }
// }

/*
 In a place, people doesn't stay at the same place for the whole time.
 The may move along the area (short or long run).
*/

void movility(Agent *agent, Agent *map[][500]) {
  int actualX = agent->posX;
  int actualY = agent->posY;

  int itsMoving = randomFloat(0.0, 1.0);

  if (itsMoving <= agent->movProba) {
    int newX = actualX;
    int newY = actualY;

    int nearMovement = randomFloat(0.0, 1.0);
    int movX = 0, movY = 0;
    int validMovement = 1;
    // Moving near
    if (nearMovement <= agent->shortDistanceMovProba) {
      do {
        validMovement = 1;
        movX = int(2 * (randomFloat(0.0, 1.0)));
        movY = int(2 * (randomFloat(0.0, 1.0)));

        if (movX > radiusMaxMovLocal) {
          movX = radiusMaxMovLocal;
        }
        if (movY > radiusMaxMovLocal) {
          movY = radiusMaxMovLocal;
        }

        if (actualX + movX >= xSize || actualX + movX < 0 ||
            actualY + movY >= ySize || actualY + movY < 0) {
          validMovement = 0;
          movX = 0, movY = 0;
        }

      } while (map[actualX + movX][actualY + movY] != NULL &&
               validMovement == 0);

    } else { // Move long distance
      do {
        validMovement = 1;
        movX = xSize * (randomFloat(0.0, 1.0));
        movY = ySize * (randomFloat(0.0, 1.0));

        if (actualX + movX >= xSize || actualX + movX < 0 ||
            actualY + movY >= ySize || actualY + movY < 0) {
          validMovement = 0;
          movX = 0, movY = 0;
        }
      } while (map[actualX + movX][actualY + movY] != NULL &&
               validMovement == 0);
    }
    newX += movX;
    newY += movY;
    map[actualX][actualY] = nullptr;

    agent->posX = newX;
    agent->posY = newY;

    map[newX][newY] = agent;
  }
}

/*
 The probability of being infected somewhere else
 the place the agents most frequent (outside the
 space we are simulating).
*/

void externContagion(Agent *agent) {
  int status = agent->status;
  if (status == 0) {
    if (randomFloat(0.0, 1.0) <= agent->extContagionProba) {
      agent->status = 1;
    }
    if (agent->status == 1) {
      contagions++;
    }
  }
}

/* This function checks the situation of all infected agents,
 *  simulating the days fighting the disease and the days before
 * the symptoms start to be present in the agent.
 */
void contagionEffects(Agent *agent) {
  if (agent->status > 0) {
    if (agent->incubationTime == 0) {
      agent->status = -1;
    }
    agent->incubationTime -= 1;

  } else if (agent->status < 0) {
    agent->recoveryTime--;
  }
}

/* Function that handles the probability of dying from
 * the disease (people alreadly infected).
 */
void decease(Agent *agent) {
  if (agent->status == -1 && agent->recoveryTime > 0) {
    if (randomFloat(0.0, 1.0) <= agent->deathProba) {
      // status == 2 ---> DEATH
      agent->status = -2;
      deaths++;
    }
  }
}

__device__ __host__ int randomInt(int limiteInferior, int limiteSuperior) {
  int randomNum;
  /* generate  number between limiteInferior and limiteSuperior: */
  randomNum = rand() % limiteSuperior + limiteInferior;
  return randomNum;
}

__device__ __host__ float randomFloat(float a, float b) {
  float random = ((float)rand()) / (float)RAND_MAX;
  float diff = b - a;
  float r = random * diff;
  return a + r;
}